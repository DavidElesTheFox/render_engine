#include "hip/hip_runtime.h"
#include <render_engine/cuda_compute/DistanceFieldKernel.h>

#include <hip/hip_runtime.h>
#include <>
#include <stdexcept>

#include <cub/device/device_radix_sort.cuh>

#include <cmath>
#include <cstdio>

using std::min;
using std::max;

#define CUDA_CHECKED_CALL(exp) {hipError_t e = (exp); assert(e == hipSuccess && #exp);}
#define ENABLE_QUERY_DEBUG false
namespace
{

    union color_t
    {
        struct
        {
            uint8_t r;
            uint8_t g;
            uint8_t b;
            uint8_t a;
        } color;
        uint32_t data;
    };
    constexpr uint32_t g_no_data_value = 0xffffffff;
    constexpr uint32_t g_max_coordinate = 4096;

    constexpr auto g_max_distance = 5000;

#pragma region Segmentation And MortonCode
    // See more details https://developer.nvidia.com/blog/thinking-parallel-part-iii-tree-construction-gpu/
    // Expands a 10-bit integer into 30 bits
    // by inserting 2 zeros after each bit.
    // A bit more details https://www.forceflow.be/2013/10/07/morton-encodingdecoding-through-bit-interleaving-implementations/
    __device__ unsigned int expandBits(unsigned int v)
    {
        v = (v * 0x00010001u) & 0xFF0000FFu;
        v = (v * 0x00000101u) & 0x0F00F00Fu;
        v = (v * 0x00000011u) & 0xC30C30C3u;
        v = (v * 0x00000005u) & 0x49249249u;
        return v;
    }

    __device__ uint3 projectToIntegerSpace(const float3& p)
    {
        return uint3{
            static_cast<uint32_t>(min(max(p.x * 1024.0f, 0.0f), 1023.0f)),
            static_cast<uint32_t>(min(max(p.y * 1024.0f, 0.0f), 1023.0f)),
            static_cast<uint32_t>(min(max(p.z * 1024.0f, 0.0f), 1023.0f))
        };
    }

    // Calculates a 30-bit Morton code for the
    // given 3D point located within the unit cube [0,1].
    __device__ unsigned int morton3D(const float3& p)
    {
        uint3 a = projectToIntegerSpace(p);
        unsigned int xx = expandBits(a.x);
        unsigned int yy = expandBits(a.y);
        unsigned int zz = expandBits(a.z);
        return xx * 4 + yy * 2 + zz;
    }
    /* See more details in Chun's page https://tmc.web.engr.illinois.edu/pub_ann.html?fbclid=IwAR26zQSewG3ohCCOJMRL7pgGDba58fo28jCuoyvirs9ZnqQUP4bjoV5dyVg
     * Shift is not used, because of the nature of the current implementation.
     * I.e.: Sort is done with a radix sort where the normalized [0,1] numbers are projected to [0, 1023]. In this way
     * the morton code can be stored in 30 bits.
     *
     * Morton code need to be stored because radix sort doesn't work with comparators but with number representations.
     *
     * Thus, any shift which is applied on the numbers on the [0,1023] range will result overflow.
     */
    __device__ bool msbIsLess(int32_t x, int32_t y)
    {
        return x < y && x < (x ^ y);
    }

    __device__ int32_t cmpShuffle(const uint3& p, const uint3 q)
    {
        auto component_at = [](const uint3& point, uint32_t idx) -> const uint32_t&
            {
                return (reinterpret_cast<const uint32_t*>(&point))[idx];
            };
        uint32_t less_idx = 0;
        uint32_t less_value = p.x ^ q.x;
        for (uint32_t k = 1; k < 3; ++k)
        {
            const uint32_t value = (component_at(p, k)) ^ (component_at(q, k));

            if (msbIsLess(less_value, value))
            {
                less_idx = k;
                less_value = value;
            }
        }
        return component_at(p, less_idx) - component_at(q, less_idx);
    }

    __global__ void segmentationKernel(hipSurfaceObject_t input,
                                       uint32_t* d_morton_codes,
                                       uint3* d_image_points,
                                       uint32_t segmentation_threshold)
    {
        const int image_width = blockDim.x * gridDim.x;
        const int image_height = blockDim.y * gridDim.y;
        const int image_depth = blockDim.z * gridDim.z;

        const uint32_t x = blockIdx.x * blockDim.x + threadIdx.x;
        const uint32_t y = blockIdx.y * blockDim.y + threadIdx.y;
        const uint32_t z = blockIdx.z * blockDim.z + threadIdx.z;

        color_t pixel;
        surf3Dread(&pixel.data, input, x * sizeof(color_t), y, z, hipBoundaryModeZero);

        const uint32_t flat_coordinate = x + y * image_width + z * (image_width * image_height);
        if (pixel.color.r != pixel.color.g
            || pixel.color.r != pixel.color.b
            || pixel.color.r < segmentation_threshold)
        {
            d_image_points[flat_coordinate] = uint3{ g_max_coordinate, g_max_coordinate, g_max_coordinate };
            d_morton_codes[flat_coordinate] = g_no_data_value;
        }
        else
        {
            const float3 normalized_coordinate{ 1.0f / static_cast<float>(image_width) * x,
                1.0f / static_cast<float>(image_height) * y,
                1.0f / static_cast<float>(image_depth) * z };
            d_image_points[flat_coordinate] = projectToIntegerSpace(normalized_coordinate);
            d_morton_codes[flat_coordinate] = morton3D(normalized_coordinate);
        }
    }
#pragma endregion

    struct AABB
    {
        uint3 min{ g_max_coordinate, g_max_coordinate, g_max_coordinate };
        uint3 max{ 0, 0, 0 };
    };

    struct DebugData
    {
        uint32_t range_begin{};
        uint32_t range_end{};
        uint32_t range_center{};
        uint3 position;
        float3 normalized_position;
        uint32_t depth{};
        bool is_point_to_debug{ false };
    };
    struct QueryData
    {
        float distance{ FLT_MAX };
        float distance_sq{ FLT_MAX };
        uint3 point;
        AABB aabb;
#if ENABLE_QUERY_DEBUG
        DebugData debug_data{};
#endif
    };

#pragma region Approximated Nearest Neighbor

#if ENABLE_QUERY_DEBUG
    __device__ void debugKernel(const QueryData& query_data, uint3* d_coordinates, const char* label = "")
    {
        if (query_data.debug_data.is_point_to_debug == false)
        {
            return;
        }

        printf("  [%d] %d--%d-->%d: {%s} Distance: %0.2f, AABB: (%d,%d,%d)->(%d, %d, %d), check: (%d, %d, %d) current best: (%d, %d, %d)\n",
               query_data.debug_data.depth,
               query_data.debug_data.range_begin,
               query_data.debug_data.range_center,
               query_data.debug_data.range_end,
               label,
               query_data.distance,
               query_data.aabb.min.x,
               query_data.aabb.min.y,
               query_data.aabb.min.z,
               query_data.aabb.max.x,
               query_data.aabb.max.y,
               query_data.aabb.max.z,
               d_coordinates[query_data.debug_data.range_center].x,
               d_coordinates[query_data.debug_data.range_center].y,
               d_coordinates[query_data.debug_data.range_center].z,
               query_data.point.x,
               query_data.point.y,
               query_data.point.z);
    }
#endif
    // comparing to the original implementation shift is not used. It is because all the points are already shifted during projection

    __device__ void check_dist(const uint3& p, const uint3& q, QueryData& query_data)
    {
        auto sq = [](uint32_t a) { return a * a; };
        float distance_sq = sq(p.x - q.x);
        distance_sq += sq(p.y - q.y);
        distance_sq += sq(p.z - q.z);
        if (distance_sq < query_data.distance_sq)
        {
            query_data.distance_sq = distance_sq;
            query_data.distance = std::sqrt(distance_sq);
            query_data.point = p;

            query_data.aabb.min.x = q.x > query_data.distance ? q.x - std::ceil(query_data.distance) : 0;
            query_data.aabb.min.y = q.y > query_data.distance ? q.y - std::ceil(query_data.distance) : 0;
            query_data.aabb.min.z = q.z > query_data.distance ? q.z - std::ceil(query_data.distance) : 0;

            query_data.aabb.max.x = q.x + query_data.distance < g_max_distance ? q.x + std::ceil(query_data.distance) : g_max_distance;
            query_data.aabb.max.y = q.y + query_data.distance < g_max_distance ? q.y + std::ceil(query_data.distance) : g_max_distance;
            query_data.aabb.max.z = q.z + query_data.distance < g_max_distance ? q.z + std::ceil(query_data.distance) : g_max_distance;
        }
    }

    // comparing to the original implementation shift is not used. It is because all the points are already shifted during projection
    __device__ float dist_sq_to_box(const uint3& q, const uint3& p1, const uint3& p2)
    {
        auto component_at = [](const uint3& point, uint32_t idx) -> const uint32_t&
            {
                return (reinterpret_cast<const uint32_t*>(&point))[idx];
            };
        auto sq = [](uint32_t a) { return a * a; };

        float less_value = 0.0f;
        for (uint32_t k = 0; k < 3; ++k)
        {
            const float value = (component_at(p1, k)) ^ (component_at(p2, k));
            if (msbIsLess(less_value, value))
            {
                less_value = value;
            }
        }
        int32_t normalization_exponent = 0;
        if (less_value != 0.0f)
        {
            frexp(less_value, &normalization_exponent);
        }
        float distance = 0.0f;
        for (uint32_t j = 0; j < 3; j++)
        {
            uint32_t p1_bottom = ((component_at(p1, j)) >> normalization_exponent) << normalization_exponent;
            uint32_t p1_up = p1_bottom + (1 << normalization_exponent);

            if (component_at(q, j) < p1_bottom)
            {
                distance += sq(component_at(q, j) - p1_bottom);
            }
            else if (component_at(q, j) > p1_up)
            {
                distance += sq(component_at(q, j) - p1_up);
            }
        }
        return distance;
    }

    __device__ void query_point(uint3* d_coordinates,
                                uint32_t range_begin,
                                uint32_t range_end,
                                uint3 point,
                                float epsilon_distance,
                                QueryData& out_result)
    {

        auto sq = [](uint32_t a) { return a * a; };

        const uint32_t range_length = range_end - range_begin;
        if (range_length == 0)
        {
            return;
        }
        const uint32_t range_center = range_begin + range_length / 2;


        check_dist(d_coordinates[range_center], point, out_result);
        if (range_length == 1
            || dist_sq_to_box(point, d_coordinates[range_begin], d_coordinates[range_end - 1]) * sq(1 + epsilon_distance) > out_result.distance_sq)
        {
            return;
        }
        if (cmpShuffle(point, d_coordinates[range_center]) < 0)
        {
            query_point(d_coordinates,
                        range_begin,
                        range_center,
                        point,
                        epsilon_distance,
                        out_result);

            if (cmpShuffle(out_result.aabb.max, d_coordinates[range_center]) > 0)
            {
                query_point(d_coordinates,
                            range_center + 1,
                            range_end,
                            point,
                            epsilon_distance,
                            out_result);
            }

        }
        else
        {
            query_point(d_coordinates,
                        range_center + 1,
                        range_end,
                        point,
                        epsilon_distance,
                        out_result);

            if (cmpShuffle(out_result.aabb.min, d_coordinates[range_center]) < 0)
            {
                query_point(d_coordinates,
                            range_begin,
                            range_center,
                            point,
                            epsilon_distance,
                            out_result);

            }
        }
    }

    __device__ void query_point_no_recursion(uint3* d_coordinates,
                                             uint32_t in_range_begin,
                                             uint32_t in_range_end,
                                             uint3 point,
                                             float epsilon_distance,
                                             QueryData& out_result)
    {
        enum class Phase
        {
            None = 0,
            CheckLowerRange_1,
            CheckUpperRange_1,
            CheckLowerRange_2,
            CheckUpperRange_2
        };
        struct SearchState
        {
            uint32_t range_begin{};
            uint32_t range_end{};
            Phase phase{ Phase::None };
        };
        constexpr uint32_t max_depth = 30;
        SearchState search_space[max_depth] = {};
        int32_t current_depth = 0;
        search_space[current_depth] = SearchState{ in_range_begin, in_range_end, Phase::None };
        auto sq = [](uint32_t a) { return a * a; };

        while (current_depth >= 0)
        {
            auto& search_data = search_space[current_depth];
            const uint32_t range_length = search_data.range_end - search_data.range_begin;

            if (range_length == 0)
            {
                current_depth--;
                continue;
            }
            const uint32_t range_center = search_data.range_begin + range_length / 2;
            if (search_data.phase == Phase::None)
            {
                check_dist(d_coordinates[range_center], point, out_result);
                if (range_length == 1
                    || dist_sq_to_box(point, d_coordinates[search_data.range_begin], d_coordinates[search_data.range_end - 1]) * sq(1 + epsilon_distance) > out_result.distance_sq)
                {
                    current_depth--;
                    continue;
                }
            }
            const int cmp_result = cmpShuffle(point, d_coordinates[range_center]);
            if (cmp_result < 0)
            {
                assert(search_data.phase != Phase::CheckLowerRange_2
                       && search_data.phase != Phase::CheckUpperRange_2);

                if (search_data.phase == Phase::None)
                {
                    search_data.phase = Phase::CheckLowerRange_1;
                }
                else if (cmpShuffle(out_result.aabb.max, d_coordinates[range_center]) > 0)
                {
                    if (search_data.phase == Phase::CheckLowerRange_1)
                    {
                        search_data.phase = Phase::CheckUpperRange_1;
                    }
                    else
                    {
                        assert(search_data.phase == Phase::CheckUpperRange_1 && "We should come to here from only the upper range calculation.");
                        search_data.phase = Phase::None;
                    }
                }
                else
                {
                    assert(search_data.phase == Phase::CheckLowerRange_1 && "We should come to here from only the lower range calculation.");
                    search_data.phase = Phase::None;
                }

            }
            else
            {
                assert(search_data.phase != Phase::CheckLowerRange_1
                       && search_data.phase != Phase::CheckUpperRange_1);

                if (search_data.phase == Phase::None)
                {
                    search_data.phase = Phase::CheckUpperRange_2;
                }
                else if (cmpShuffle(out_result.aabb.min, d_coordinates[range_center]) < 0)
                {
                    if (search_data.phase == Phase::CheckUpperRange_2)
                    {
                        search_data.phase = Phase::CheckLowerRange_2;
                    }
                    else
                    {
                        assert(search_data.phase == Phase::CheckLowerRange_2 && "We should come to here from only the lower range calculation.");

                        search_data.phase = Phase::None;
                    }
                }
                else
                {
                    assert(search_data.phase == Phase::CheckUpperRange_2 && "We should come to here from only the upper range calculation.");
                    search_data.phase = Phase::None;
                }
            }

            switch (search_data.phase)
            {
                case Phase::None:
                    current_depth--;
                    break;
                case Phase::CheckUpperRange_1:
                case Phase::CheckUpperRange_2:
                    current_depth++;
                    assert(current_depth < max_depth);
                    search_space[current_depth] = {
                        range_center + 1,
                        search_data.range_end,
                        Phase::None
                    };
                    break;
                case Phase::CheckLowerRange_1:
                case Phase::CheckLowerRange_2:
                    current_depth++;
                    assert(current_depth < max_depth);
                    search_space[current_depth] = {
                        search_data.range_begin,
                        range_center,
                        Phase::None
                    };
                    break;
                default:
                    break;
            }
        }
    }

    __global__ void
        __launch_bounds__(288, 8)
        distanceFieldKernel(uint3* d_coordinates,
                            uint32_t point_count,
                            float epsilon_distance,
                            hipSurfaceObject_t output)
    {
        const int image_width = blockDim.x * gridDim.x;
        const int image_height = blockDim.y * gridDim.y;
        const int image_depth = blockDim.z * gridDim.z;

        const uint32_t x = blockIdx.x * blockDim.x + threadIdx.x;
        const uint32_t y = blockIdx.y * blockDim.y + threadIdx.y;
        const uint32_t z = blockIdx.z * blockDim.z + threadIdx.z;

        const float3 normalized_p{ x / static_cast<float>(image_width),
        y / static_cast<float>(image_height),
        z / static_cast<float>(image_depth) };
        QueryData result{};
        uint3 p = projectToIntegerSpace(normalized_p);


        query_point_no_recursion(d_coordinates,
                                 0,
                                 point_count,
                                 p,
                                 epsilon_distance,
                                 result);
        surf3Dwrite(result.distance / 1023.0f, output, x * 4, y, z, hipBoundaryModeZero);
    }
#pragma endregion
}

namespace RenderEngine
{
    namespace CudaCompute
    {

        hipArray_t DistanceFieldKernel::allocateInputMemory(uint32_t width, uint32_t height, uint32_t depth)
        {

            hipArray_t device_data{ nullptr };
            hipChannelFormatDesc channel_format = hipCreateChannelDesc(8, 8, 8, 8, hipChannelFormatKindUnsigned);
            CUDA_CHECKED_CALL(hipMalloc3DArray(&device_data,
                                                &channel_format,
                                                hipExtent{ width, height, depth },
                                                hipArraySurfaceLoadStore));

            return device_data;
        }
        hipArray_t DistanceFieldKernel::allocateOutputMemory(uint32_t width, uint32_t height, uint32_t depth)
        {
            hipArray_t device_data{ nullptr };
            hipChannelFormatDesc channel_format = hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindFloat);

            CUDA_CHECKED_CALL(hipMalloc3DArray(&device_data,
                                                &channel_format,
                                                hipExtent{ width, height, depth },
                                                hipArraySurfaceLoadStore));

            return device_data;
        }
        void DistanceFieldKernel::freeMemory(hipArray_t device_memory)
        {
            CUDA_CHECKED_CALL(hipFreeArray(device_memory));
        }
        DistanceFieldKernel::DistanceFieldKernel(const KernelParameters& kernel_parameters,
                                                 hipStream_t hip_stream)
            : _kernel_parameters(kernel_parameters)
            , _cuda_stream(hip_stream)
        {}

        void DistanceFieldKernel::run(const hipSurfaceObject_t d_input_data,
                                      hipSurfaceObject_t d_output_data,
                                      uint32_t segmentation_threshold,
                                      float epsilon_distance)
        {
            const uint32_t width = _kernel_parameters.block_size.x * _kernel_parameters.grid_size.x;
            const uint32_t height = _kernel_parameters.block_size.y * _kernel_parameters.grid_size.y;
            const uint32_t depth = _kernel_parameters.block_size.z * _kernel_parameters.grid_size.z;
            uint32_t num_items = width * height * depth;



            uint32_t* d_kd_tree_morton_codes{ nullptr };
            CUDA_CHECKED_CALL(hipMalloc(&d_kd_tree_morton_codes, num_items * sizeof(uint32_t)));
            uint3* d_kd_tree_coordinates{ nullptr };
            CUDA_CHECKED_CALL(hipMalloc(&d_kd_tree_coordinates, num_items * sizeof(uint3)));
            segmentationKernel << <_kernel_parameters.grid_size, _kernel_parameters.block_size, 0, _cuda_stream >> > (
                d_input_data,
                d_kd_tree_morton_codes,
                d_kd_tree_coordinates,
                segmentation_threshold);

            uint32_t* d_kd_tree_morton_codes_sorted{ nullptr };
            CUDA_CHECKED_CALL(hipMalloc(&d_kd_tree_morton_codes_sorted, num_items * sizeof(uint3)));
            uint3* d_kd_tree_coordinates_sorted{ nullptr };
            CUDA_CHECKED_CALL(hipMalloc(&d_kd_tree_coordinates_sorted, num_items * sizeof(uint3)));
            hipcub::DeviceRadixSort radix_sort;
            size_t temporary_size = 0;
            uint32_t* d_temporary_memory = nullptr;
            CUDA_CHECKED_CALL(radix_sort.SortPairs(nullptr,
                                                   temporary_size,
                                                   d_kd_tree_morton_codes,
                                                   d_kd_tree_morton_codes_sorted,
                                                   d_kd_tree_coordinates,
                                                   d_kd_tree_coordinates_sorted,
                                                   num_items));

            CUDA_CHECKED_CALL(hipMalloc(&d_temporary_memory, temporary_size));

            CUDA_CHECKED_CALL(radix_sort.SortPairs(d_temporary_memory,
                                                   temporary_size,
                                                   d_kd_tree_morton_codes,
                                                   d_kd_tree_morton_codes_sorted,
                                                   d_kd_tree_coordinates,
                                                   d_kd_tree_coordinates_sorted,
                                                   num_items,
                                                   0,
                                                   sizeof(uint32_t) * 8,
                                                   _cuda_stream));


            std::vector<uint32_t> kd_tree_morton_codes_sorted(num_items, uint32_t{});
            CUDA_CHECKED_CALL(hipMemcpy(kd_tree_morton_codes_sorted.data(), d_kd_tree_morton_codes_sorted, num_items * sizeof(uint32_t), hipMemcpyDeviceToHost));

            const uint32_t num_of_zeros_at_end = std::find_if(kd_tree_morton_codes_sorted.rbegin(), kd_tree_morton_codes_sorted.rend(),
                                                              [](uint32_t morton_code) { return morton_code != g_no_data_value; })
                - kd_tree_morton_codes_sorted.rbegin();


            CUDA_CHECKED_CALL(hipFree(d_temporary_memory));
            CUDA_CHECKED_CALL(hipFree(d_kd_tree_coordinates));
            CUDA_CHECKED_CALL(hipFree(d_kd_tree_morton_codes));
            CUDA_CHECKED_CALL(hipFree(d_kd_tree_morton_codes_sorted));


            distanceFieldKernel << <_kernel_parameters.grid_size, _kernel_parameters.block_size, 0, _cuda_stream >> > (
                d_kd_tree_coordinates_sorted,
                num_items - num_of_zeros_at_end,
                epsilon_distance,
                d_output_data);
            CUDA_CHECKED_CALL(hipFree(d_kd_tree_coordinates_sorted));

        }
    }
}